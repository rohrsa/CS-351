
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

using Count = size_t;
using DataType = long;

const DataType DefalutStartValue = -6.0;
const Count TestSize = 1'000'000'000;
const Count NumCheckValues = 500;

//
// --- Add your CUDA kernel implementation of iota here
//

int main(int argc, char* argv[]) {
    Count numValues = argc > 1 ? std::stol(argv[1]) : TestSize;

    std::vector<DataType> values(numValues);

    size_t numBytes = numValues * sizeof(DataType);

    DataType* gpuValues;
    hipMalloc(&gpuValues, numBytes);
  
    hipMemcpy(gpuValues, values.data(), numBytes,hipMemcpyHostToDevice);

    DataType startValue = DefalutStartValue;

    int chunkSize = 256;
    int numChunks = int((float) numValues / chunkSize + 1);
    iota<<<numChunks, chunkSize>>>(numValues, gpuValues, startValue);
  
    hipMemcpy(values.data(), gpuValues, numBytes, hipMemcpyDeviceToHost);

    Count step = numValues / NumCheckValues;
    for (int i = 6, n = 0; i < numValues && n < NumCheckValues; ++n, i += step) {
        DataType checkValue = startValue + static_cast<DataType>(i);

        if (values[i] != checkValue) {
            std::cerr << "Values do not match for position " << i
                << values[i] << " != " << checkValue << "\n";
            exit(EXIT_FAILURE);
        }
    }
}
