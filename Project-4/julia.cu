#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
//
//  julia.cu - A CUDA-based Julia set generator
//

#include <complex>
#include <fstream>
#include <iostream>
#include <vector>

#define DEBUG
#include "CudaCheck.h"

//----------------------------------------------------------------------------
//
//  Global configuration parameters
//

const size_t Width = 1024;
const size_t Height = 1024;
const size_t MaxIterations = 1000;

//----------------------------------------------------------------------------
//
//  struct Color
//
//  A simple struture for storing a color as an RGB triple.  In this case,
//    the color components are stored as unsigned chars, which is what's
//    used in the PPM format.
//
//  This structure is used both on the CPU and the GPU, and so the
//    constructor is tagged to have a version of the function generated for
//    both the CPU (signified by the __host__ decoration), and for the
//    GPU (__device__).
//

struct Color {
    using T = unsigned char;
    T r = 0;
    T g = 0;
    T b = 0;
    
    __host__ __device__
    Color() { /* Empty */ }

    __host__ __device__ 
    Color(T r, T g, T b) : r(r), g(g), b(b) { /* Empty */}
};

//----------------------------------------------------------------------------
//
//  function setColor()
//
//  This function returns a color based on an iteration value.
//
//  As this function is called from an executing GPU kernel (julia), it's
//    tagged as being a device-only function (using the __device__ decoration)
//

__device__
Color setColor(int iterations) {
    constexpr size_t NumColors = 16;
    const Color colors[NumColors] = {
        Color(66, 30, 15),
        Color(25, 7, 26),
        Color(9, 1, 47),
        Color(4, 4, 73),
        Color(0, 7, 100),
        Color(12, 44, 138),
        Color(24, 82, 177),
        Color(57, 125, 209),
        Color(134, 181, 229),
        Color(211, 236, 248),
        Color(241, 233, 191),
        Color(248, 201, 95),
        Color(255, 170, 0),
        Color(204, 128, 0),
        Color(153, 87, 0),
        Color(106, 52, 3)
    };

    const Color black;

    return iterations < MaxIterations ? colors[iterations % NumColors] : black;
}

//----------------------------------------------------------------------------
//
//  template struct TComplex (for complex numbers)
//
//  A proper C++ class (a struct is merely a C++ class with the default
//    member set to "public", where a class is "private" by default) for
//    storing a complex value.  In this case, we use 'x' as the real
//    component, and 'y' for the imaginary component.
//
//  For the purposes of the program, we use a typedef (using a modern
//    C++ using statement) to define a float-based complex value.
//
//  More interesting is that a number of the class methods are decorated
//    with __host__ and __device__ depending on their use in the program.
//    For example, the constructor is used both on the CPU, and in the GPU
//    kernel.  As such, that function needs to be decorated with both options.
//    By way of comparison, the function magnitude() is only used (in this
//    case) in the julia device kernel, so it's tagged as __device__.
//
//  Any function that isn't decorated is compiled for the CPU only by default.
//

template <typename T>
struct TComplex {
    T x = T(0);
    T y = T(0);

    TComplex() = default;

    __host__ __device__
    TComplex(T x, T y) : x(x), y(y) {}

    __device__
    T magnitude() const 
        { return sqrt(x*x + y*y); }

    __device__
    TComplex& operator -= (const TComplex& a)
        { x -= a.x; y -= a.y; return *this; }

    __device__ __host__
    friend TComplex operator + (const TComplex& a, const TComplex& b)
        { return TComplex(a.x + b.x, a.y + b.y); }

    __device__
    friend TComplex operator * (const TComplex& a, const TComplex& b)
        { return TComplex(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x); }

    friend TComplex operator - (const TComplex& a, const TComplex& b)
        { return TComplex(a.x - b.x, a.y - b.y); }

    friend TComplex operator * (const float s, const TComplex& a)
        { return TComplex(s * a.x, s * a.y); }

    friend std::ostream& operator << (std::ostream& os, const TComplex& p)
        { return os << "(" << p.x << ", " << p.y << ")"; }
};

using Complex = TComplex<float>;

//----------------------------------------------------------------------------
//
//  compute kernel julia
//
//  A Julia set generator kernel, which iterates complex values to determine
//    if they converge, or not.
//
//  The parameters for the kernel are:
//
//  - d, which is declared as a complex number is really just conviently
//      storing the "pixel size" in its value.  The values in 'd' are used
//      to determine which coordinate in the complex plane we're working
//      with.
//
//  - center, which is the complex coordinate of the center of our Julia
//      set.  The points we use in our iteration computations are centered
//      around this center value.
//
//  - pixels, which is the "image", a collection of Color structures that
//      will be written out to form our PPM image.
//
//  This CUDA kernel is tagged __global__, as it's executed on the GPU, but
//    launched (dispatched) from the CPU.
//

inline __device__ float magnitude(const Complex& z) { return z.magnitude(); }

__global__
void julia(Complex d, Complex center, Color* pixels) {
    // Add your CUDA implementation of the Julia program here.
    //
    // Hint: this function should basically be the same thing as the body
    //   of the two for loops in the C++ version.  If you're clever, which
    //   means you choose your variable names well (just like Phil mentions)
    //   you can pretty much drop in the CPU code, and then do the extra
    //   CUDA bits
}

//----------------------------------------------------------------------------
//
//  function main()
//
//  Nothing particularly special here.  We specify the domain in the Complex
//    plane we're interested in looking at using its lower-left, and upper-
//    right corners (the variables "ll", and "ur", respectively).
//
//  From there, we determine the size of the domain, its center point, and
//    the size of a pixel in the complex plane.
//
//  Then we allocate some GPU memory, specify the number of threads we
//    want to execute in parallel (which we set in the "blockDim" variable).
//    Using the block's dimensions, we specify how many blocks are required
//    in each of the dimensions to populate our image, and then we dispatch
//    our kernel to do its work.
//
//  For our CUDA-specific operations, we verify their proper operation
//    using some modifications of our CUDA checking macros.  Of particular
//    note is for the CUDA_CHECK_KERNEL() macro, the kernel dispatch needs
//    to be wrapped in an extra set of parentheses to trick the C preprocessor
//    that there's only a single parameter (the comma in the kernel's 
//    dispatch messes things up).
//
//  Once the kernel's done, we copy the results from the GPU back to the
//    CPU, and output the results in an image named "julia.ppm".
//

int main() {
    Complex ll(-2.1, -2.1);
    Complex ur( 2.1,  2.1);
    Complex domain = ur - ll;
    Complex center = 0.5 * domain;
    Complex d(domain.x/Width, domain.y/Height);

    Color* gpuPixels;
    size_t numBytes = Width * Height * sizeof(Color);
    CUDA_CHECK_CALL(hipMalloc(&gpuPixels, numBytes));

    dim3 blockDim(32, 32);
    dim3 numBlocks(Width/blockDim.x, Height/blockDim.y);
    CUDA_CHECK_KERNEL( (julia<<<numBlocks, blockDim>>>(d, center, gpuPixels)) );

    Color* pixels = new Color[Width * Height];
    CUDA_CHECK_CALL(hipMemcpy(pixels, gpuPixels, numBytes, hipMemcpyDeviceToHost));

    std::ofstream ppm("julia.ppm", std::ios::binary);
    ppm << "P6 " << Width << " " << Height << " " << 255 << "\n";
    ppm.write(reinterpret_cast<const char*>(&pixels[0]), numBytes);
}
